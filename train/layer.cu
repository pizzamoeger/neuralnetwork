#include "hip/hip_runtime.h"
#include "includes.h"

std::random_device rd;
std::default_random_engine generator(rd());

void fully_connected_layer::init(layer_data data, layer_data data_previous, float* new_delta) {

    data.n_in = {data_previous.n_out.feature_maps * data_previous.n_out.y * data_previous.n_out.x, 1, 1};
    data.elems = data.n_in.x+data_previous.elems;
    this->data = data;

    hipMalloc((void**) &delta, data.n_out.x*sizeof(float));
    this->new_delta = new_delta;

    hipMalloc((void**) &this->dev_data, sizeof(layer_data));
    hipMalloc((void**) &this->dev_data_previous, sizeof(layer_data));

    hipMemcpy(this->dev_data, &data, sizeof(layer_data), hipMemcpyHostToDevice);
    hipMemcpy(this->dev_data_previous, &data_previous, sizeof(layer_data), hipMemcpyHostToDevice);

    hipMalloc((void**) &dev_weights, data.n_out.x*data.n_in.x*sizeof(float));
    hipMalloc((void**) &dev_weights_vel, data.n_out.x*data.n_in.x*sizeof(float));
    hipMalloc((void**) &dev_weights_updt, data.n_out.x*data.n_in.x*sizeof(float));

    // weights init: https://www.analyticsvidhya.com/blog/2021/05/how-to-initialize-weights-in-neural-networks/
    // https://wandb.ai/sauravmaheshkar/initialization/reports/A-Gentle-Introduction-To-Weight-Initialization-for-Neural-Networks--Vmlldzo2ODExMTg
    // https://stats.stackexchange.com/questions/373136/softmax-weights-initialization
    float stddev;
    if (data.activation_function == RELU) stddev = sqrt(2.0/data.n_in.x); // He-et-al
    else stddev = sqrt(2.0/data.n_in.x+data.n_in.x); // Xavier
    float* dev_stddev;
    hipMalloc((void**) &dev_stddev, sizeof(float));
    hipMemcpy(dev_stddev, &stddev, sizeof(float), hipMemcpyHostToDevice);
    set_to_random<<<data.n_out.x * data.n_in.x, 1>>>(dev_weights, dev_stddev);
    set_to<<<data.n_out.x * data.n_in.x, 1>>>(dev_weights_vel, 0);
    set_to<<<data.n_out.x * data.n_in.x, 1>>>(dev_weights_updt, 0);

    hipMalloc((void**) &dev_biases, data.n_out.x*sizeof(float));
    hipMalloc((void**) &dev_biases_vel, data.n_out.x*sizeof(float));
    hipMalloc((void**) &dev_biases_updt, data.n_out.x*sizeof(float));
    // biases init: https://medium.com/@glenmeyerowitz/bias-initialization-in-a-neural-network-2e5d26fed0f0
    set_to<<<data.n_out.x, 1>>>(dev_biases, 0.01);
    set_to<<<data.n_out.x,1>>>(dev_biases_vel, 0);
    set_to<<<data.n_out.x,1>>>(dev_biases_updt, 0);

    hipFree(dev_stddev);
}

void fully_connected_layer::feedforward(float* dev_a, float* dev_dz) {
/*
    if (data.activation_function == SOFTMAX) {
        // TODO: make this smart
        reduce<<<data.n_out.x, data.n_in.x, data.n_in.x*sizeof(float)>>>(dev_weights, &dev_a[data.elems], &dev_data->n_in.x, &dev_data->n_in.x, CALC_Z, &dev_a[data.elems-data.n_in.x], dev_biases);
        hipDeviceSynchronize();

        float* exp_vec;
        float* sum_of_exp;
        hipMalloc((void**) &exp_vec, data.n_out.x*sizeof(float));
        hipMalloc((void**) &sum_of_exp, sizeof(float));
        set_to<<<1,1>>> (sum_of_exp, 0);
        hipDeviceSynchronize();
        //assert(data.n_out.x < (1<<10));

        int *max_id;
        hipMalloc((void**) &max_id, sizeof(int));
        find_max<<<1,1>>>(&dev_a[data.elems], max_id, &dev_data->n_out.x);
        calc_exp<<<data.n_out.x, 1>>>(exp_vec, &dev_a[data.elems], max_id); // this could also be done in the reduce func
        hipDeviceSynchronize();

        reduce<<<1, data.n_out.x, data.n_out.x*sizeof(float)>>>(exp_vec, sum_of_exp, &dev_data->n_out.x, &dev_data->n_out.x, ADD_EXP);
        hipDeviceSynchronize();

        calc_a_and_dz<<<data.n_out.x, 1>>>(&dev_a[data.elems], &dev_dz[data.elems], &dev_data->activation_function, sum_of_exp);
        hipDeviceSynchronize();

        hipFree(max_id);
        hipFree(exp_vec);
        hipFree(sum_of_exp);
        //hipDeviceSynchronize();
    } else {*/
        //reduce<<<data.n_out.x, data.n_in.x, data.n_in.x*sizeof(float)>>>(dev_weights, &dev_a[data.elems], &dev_data->n_in.x, &dev_data->n_in.x, CALC_Z, &dev_a[data.elems-data.n_in.x], dev_biases, &dev_dz[data.elems], &dev_data->activation_function
        reduce<<<data.n_out.x, (1<<10), data.n_in.x*sizeof(float)>>>(dev_weights, &dev_a[data.elems], &dev_data->n_in.x, &dev_data->n_in.x, CALC_Z, &dev_a[data.elems-data.n_in.x], dev_biases, &dev_dz[data.elems], &dev_data->activation_function);
    hipDeviceSynchronize();
    //}
}

void fully_connected_layer::backprop(float* activations, float* derivative_z) {
    // TODO: this could be made faster but also uglier
    backprop_logic<<<data.n_out.x,data.n_in.x>>>(dev_weights_updt, delta, &activations[data.elems-data.n_in.x], dev_biases_updt, &dev_data->n_in.x);
    reduce<<<data.n_in.x, (1<<10), data.n_out.x*sizeof(float)>>>(dev_weights, new_delta, &dev_data->n_out.x, &dev_data->n_out.x, CALC_ND, delta, &derivative_z[data.elems-data.n_in.x]);
    hipDeviceSynchronize();
}

void fully_connected_layer::update(hyperparams* dev_params) {
    // update velocities
    ::update<<<data.n_out.x, data.n_in.x>>> (dev_biases_vel, dev_weights_vel, dev_weights_updt, dev_biases_updt, dev_weights, dev_biases, dev_params);
    hipDeviceSynchronize();
}

void fully_connected_layer::save(std::string filename) {
    std::ofstream file(filename, std::ios_base::app);

    file << LAYER_NUM_FULLY_CONNECTED << "//";
    file << data.activation_function << "//";
    file << data.n_out.x << "//";

    float* biases = new float [data.n_out.x];
    hipMemcpy(biases, dev_biases, data.n_out.x*sizeof(float), hipMemcpyDeviceToHost);
    for (int bias = 0; bias < data.n_out.x; bias++) file << biases[bias] << " ";
    delete[] biases;
    file << "//";

    float* biases_vel = new float [data.n_out.x];
    hipMemcpy(biases_vel, dev_biases_vel, data.n_out.x*sizeof(float), hipMemcpyDeviceToHost);
    for (int bias_vel = 0; bias_vel < data.n_out.x; bias_vel++) file << biases_vel[bias_vel] << " ";
    delete[] biases_vel;
    file << "//";

    float* weights = new float [data.n_out.x*data.n_in.x];
    hipMemcpy(weights, dev_weights, data.n_out.x*data.n_in.x*sizeof(float), hipMemcpyDeviceToHost);
    for (int weight = 0; weight < data.n_out.x*data.n_in.x; weight++) file << weights[weight] << " ";
    delete[] weights;
    file << "//";

    float* weights_vel = new float [data.n_out.x*data.n_in.x];
    hipMemcpy(weights_vel, dev_weights_vel, data.n_out.x*data.n_in.x*sizeof(float), hipMemcpyDeviceToHost);
    for (int weight = 0; weight < data.n_out.x*data.n_in.x; weight++) file << weights_vel[weight] << " ";
    delete[] weights_vel;
    file << "\n";

    file.close();
}

void fully_connected_layer::clear() {
    hipFree(delta);
    hipFree(dev_weights);
    hipFree(dev_weights_vel);
    hipFree(dev_weights_updt);
    hipFree(dev_biases);
    hipFree(dev_biases_vel);
    hipFree(dev_biases_updt);
    hipFree(dev_data_previous);
    hipFree(dev_data);
}

/*void convolutional_layer::init(layer_data data, layer_data data_previous) {

    data.n_in = data_previous.n_out;
    data.n_out.x = (data.n_in.x - data.receptive_field_length + 1) / data.stride_length;
    data.n_out.y = (data.n_in.y - data.receptive_field_length + 1) / data.stride_length;

    this->data = data;
    this->data_previous = data_previous;

    weights_size = data.n_in.feature_maps * data.n_out.feature_maps * data.receptive_field_length * data.receptive_field_length;

    normal_distribution<float> distribution(0.0, 1.0 / sqrt(data.receptive_field_length * data.receptive_field_length));

    biases = new float[data.n_out.feature_maps];
    biasesVelocity = new float[data.n_out.feature_maps];
    for (int map = 0; map < data.n_out.feature_maps; map++) {
        biases[map] = distribution(generator);
        biasesVelocity[map] = 0;
    }

    weights = new float[weights_size];
    weightsVelocity = new float[weights_size];
    for (int previous_map = 0; previous_map < data.n_in.feature_maps; previous_map++) {
        for (int map = 0; map < data.n_out.feature_maps; map++) {
            for (int kernel_y = 0; kernel_y < data.receptive_field_length; kernel_y++) {
                for (int kernel_x = 0; kernel_x < data.receptive_field_length; kernel_x++) {
                    weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] = distribution(generator);
                    weightsVelocity[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] = 0;
                }
            }
        }
    }

    updateB = new float[data.n_out.feature_maps];
    updateW = new float[weights_size];
    for (int bias = 0; bias < data.n_out.feature_maps; bias++) updateB[bias] = 0;
    for (int weight = 0; weight < weights_size; weight++) updateW[weight] = 0;
}

void convolutional_layer::feedforward(float* a, float* dz, float* &new_a, float* &new_dz) {
    (void) dz;

    float* z = new float [data.n_out.feature_maps * data.n_out.y * data.n_out.x];
    for (int i = 0; i < data.n_out.feature_maps * data.n_out.y * data.n_out.x; i++) z[i] = 0;

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) {
                for (int previous_map = 0; previous_map < data.n_in.feature_maps; previous_map++) {
                    for (int kernel_y = 0; kernel_y < data.receptive_field_length; kernel_y++) {
                        for (int kernel_x = 0; kernel_x < data.receptive_field_length; kernel_x++) {
                            z[get_data_index(map, y, x, data)] +=
                                    weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] *
                                    a[get_data_index(previous_map, y * data.stride_length + kernel_y, x * data.stride_length + kernel_x, data_previous)];
                        }
                    }
                }
                z[get_data_index(map, y, x, data)] += biases[map];
                new_a[get_data_index(map, y, x, data)] = activation_function(z[get_data_index(map, y, x, data)], data.activation_function);
                new_dz[get_data_index(map, y, x, data)] = activation_function_prime(z[get_data_index(map, y, x, data)], data.activation_function);
            }
        }
    }

    delete[] z;
}

void convolutional_layer::backprop(float * &delta,
                                   float* &activations,
                                   float* &derivative_z, float * &new_delta) {

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) delta[get_data_index(map, y, x, data)] *= derivative_z[get_data_index(map, y, x, data)];
        }
    }

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) {
                updateB[map] += delta[get_data_index(map, y, x, data)];
                for (int previous_map = 0; previous_map < data.n_in.feature_maps; previous_map++) {
                    for (int kernel_y = 0; kernel_y < data.receptive_field_length; kernel_y++) {
                        for (int kernel_x = 0; kernel_x < data.receptive_field_length; kernel_x++) {
                            new_delta[get_data_index(previous_map, y * data.stride_length + kernel_y, x * data.stride_length +
                                                                                      kernel_x, data_previous)] +=
                                    delta[get_data_index(map, y, x, data)] * weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)];
                            updateW[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] +=
                                    activations[get_data_index(previous_map, y * data.stride_length + kernel_y,
                                            x * data.stride_length + kernel_x, data)] * delta[get_data_index(map, y, x, data)];
                        }
                    }
                }
            }
        }
    }
}

void convolutional_layer::update(hyperparams params) {

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        biasesVelocity[map] = params.momentum_coefficient * biasesVelocity[map] -
                              (params.convolutional_biases_learning_rate / params.mini_batch_size) * updateB[map];
        biases[map] += biasesVelocity[map];
    }

    for (int previous_map = 0; previous_map < data.n_in.feature_maps; previous_map++) {
        for (int map = 0; map < data.n_out.feature_maps; map++) {
            for (int kernel_y = 0; kernel_y < data.receptive_field_length; kernel_y++) {
                for (int kernel_x = 0; kernel_x < data.receptive_field_length; kernel_x++) {
                    weightsVelocity[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] =
                            params.momentum_coefficient * weightsVelocity[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] -
                            (params.convolutional_weights_learning_rate / params.mini_batch_size /
                             (data.n_out.x * data.n_out.y) *
                             data.stride_length * data.stride_length) * updateW[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)];
                    weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] = (1 -
                                                                      params.convolutional_weights_learning_rate /
                                                                      (data.n_out.x * data.n_out.y) *
                                                                      data.stride_length * data.stride_length *
                                                                      params.L2_regularization_term /
                                                                      params.training_data_size) *
                                                                     weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] +
                                                                     weightsVelocity[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)];
                }
            }
        }
    }

    for (int i = 0; i < data.n_out.feature_maps; i++) updateB[i] = 0;
    for (int i = 0; i < data.n_in.feature_maps * data.n_out.feature_maps * data.receptive_field_length * data.receptive_field_length; i++) updateW[i] = 0;
}

void convolutional_layer::save(string filename) {
    ofstream file(filename, std::ios_base::app);

    file << LAYER_NUM_CONVOLUTIONAL << "//";
    file << data.activation_function << "//";
    file << data.stride_length << " " << data.receptive_field_length << " " << data.n_out.feature_maps << "//";

    for (int bias = 0; bias < data.n_out.feature_maps; bias++) file << biases[bias] << " ";
    file << "//";
    for (int biasVel = 0; biasVel < data.n_out.feature_maps; biasVel++) file << biasesVelocity[biasVel] << " ";
    file << "//";
    for (int weight = 0; weight < weights_size; weight++) file << weights[weight] << " ";
    file << "//";
    for (int weightVel = 0; weightVel < weights_size; weightVel++) file << weightsVelocity[weightVel] << " ";
    file << "\n";

    file.close();
}

void convolutional_layer::clear() {
    delete[] weights;
    delete[] weightsVelocity;
    delete[] biases;
    delete[] biasesVelocity;
    delete[] updateW;
    delete[] updateB;
}

void max_pooling_layer::init(layer_data data, layer_data data_previous) {
    data.n_in = data_previous.n_out;
    this->data = data;
    this->data_previous = data_previous;
    this->data.n_out.x = data.n_in.x / data.summarized_region_length;
    this->data.n_out.y = data.n_in.y / data.summarized_region_length;
    this->data.n_out.feature_maps = data_previous.n_out.feature_maps;
}

void max_pooling_layer::feedforward(float* a, float* dz, float* &new_a, float* &new_dz) {
    (void) dz;

    for (int i = 0; i < data.n_out.feature_maps * data.n_out.y * data.n_out.x; i++) new_a[i] = numeric_limits<float>::lowest();

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) {
                for (int kernel_y = 0; kernel_y < data.summarized_region_length; kernel_y++) {
                    for (int kernel_x = 0; kernel_x < data.summarized_region_length; kernel_x++) {
                        new_a[get_data_index(map, y, x, data)] = max(new_a[get_data_index(map, y, x, data)], a[get_data_index(map, y * data.summarized_region_length + kernel_y, x * data.summarized_region_length + kernel_x, data_previous)]);
                    }
                }
                new_dz[get_data_index(map, y, x, data)] = new_a[get_data_index(map, y, x, data)];
            }
        }
    }
}

void max_pooling_layer::backprop(float * &delta,
                                 float* &activations, float* &derivative_z, float * &new_delta) {
    const float epsilon = 1e-8;

    //cout << activations[get_data_index(data.n_out.feature_maps-1, (data.n_out.y-1)*data.summarized_region_length+data.summarized_region_length-1, (data.n_out.x-1)*data.summarized_region_length+data.summarized_region_length-1, data_previous)] << "sdfkjdslksfjlsf\n";
    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) {
                for (int kernel_y = 0; kernel_y < data.summarized_region_length; kernel_y++) {
                    for (int kernel_x = 0; kernel_x < data.summarized_region_length; kernel_x++) {
                        int act = activations[get_data_index(map, y * data.summarized_region_length + kernel_y, x * data.summarized_region_length + kernel_x, data_previous)];
                        int dev = derivative_z[get_data_index(map, y, x, data)];
                        if (act < dev) swap(act, dev);
                        if (act - dev < epsilon) {
                            new_delta[get_data_index(map, y * data.summarized_region_length + kernel_y,
                                    x * data.summarized_region_length + kernel_x, data_previous)] = delta[get_data_index(map, y, x, data)];
                        }
                    }
                }
            }
        }
    }
}

void max_pooling_layer::update(hyperparams params) {
    (void) params;
}

void max_pooling_layer::save(string filename) {
    ofstream file(filename, std::ios_base::app);

    file << LAYER_NUM_MAX_POOLING << "//";
    file << data.summarized_region_length << "\n";

    file.close();
}

void max_pooling_layer::clear() {}
*/
void input_layer::init(layer_data data, layer_data data_previous, float* new_delta) {
    data.elems = 0;
    this->data = data;
    hipMalloc((void**) &delta, data.n_out.feature_maps*data.n_out.y*data.n_out.x*sizeof(float));
    (void) data_previous;
}

void input_layer::feedforward(float* a, float* dz) {}

void input_layer::backprop(float* activations, float* derivative_z) {}

void input_layer::update(hyperparams* params) {}

void input_layer::save(std::string filename) {
    std::ofstream file(filename, std::ios_base::app);

    file << LAYER_NUM_INPUT << "//";
    file << data.n_out.x << " " << data.n_out.y << "\n";

    file.close();
}

void input_layer::clear() {
    hipFree(delta);
}
