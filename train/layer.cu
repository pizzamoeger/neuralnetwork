#include "hip/hip_runtime.h"
#include "includes.h"

random_device rd;
default_random_engine generator(rd());

void fully_connected_layer::init(layer_data data, layer_data data_previous) {

    data.n_in = {data_previous.n_out.feature_maps * data_previous.n_out.y * data_previous.n_out.x, 1, 1};
    this->data = data;
    this->data_previous = data_previous;

    hipMalloc((void**) &this->dev_data, sizeof(layer_data));
    hipMalloc((void**) &this->dev_data_previous, sizeof(layer_data));

    hipMemcpy(this->dev_data, &data, sizeof(layer_data), hipMemcpyHostToDevice);
    hipMemcpy(this->dev_data_previous, &data_previous, sizeof(layer_data), hipMemcpyHostToDevice);

    hipMalloc((void**) &dev_weights, data.n_out.x*data.n_in.x*sizeof(float));
    hipMalloc((void**) &dev_weights_vel, data.n_out.x*data.n_in.x*sizeof(float));
    hipMalloc((void**) &dev_weights_updt, data.n_out.x*data.n_in.x*sizeof(float));
    set_to_random<<<data.n_out.x * data.n_in.x, 1>>>(dev_weights, &this->dev_data->n_in.x);
    set_to<<<data.n_out.x * data.n_in.x, 1>>>(dev_weights_vel, 0);
    set_to<<<data.n_out.x * data.n_in.x, 1>>>(dev_weights_updt, 0);

    hipMalloc((void**) &dev_biases, data.n_out.x*sizeof(float));
    hipMalloc((void**) &dev_biases_vel, data.n_out.x*sizeof(float));
    hipMalloc((void**) &dev_biases_updt, data.n_out.x*sizeof(float));
    set_to_random<<<data.n_out.x, 1>>>(dev_biases, &this->dev_data->n_in.x);
    set_to<<<data.n_out.x,1>>>(dev_biases_vel, 0);
    set_to<<<data.n_out.x,1>>>(dev_biases_updt, 0);
}

void fully_connected_layer::feedforward(float* &dev_a, float* &dev_dz, float* &dev_z, int* dev_elems) {
    calc_z<<<data.n_out.x, data.n_in.x>>>(dev_a, dev_weights, dev_biases, dev_z, &dev_data->n_in.x, dev_elems);
    hipDeviceSynchronize();
    float* sum_of_exp;
    hipMalloc((void**) &sum_of_exp, sizeof(float));
    set_to<<<1,1>>> (sum_of_exp, 0);
    if (data.activation_function == SOFTMAX) calc_sum_of_exp<<<data.n_out.x, 1>>>(sum_of_exp, dev_z, dev_elems);
    /*float sum;
    hipMemcpy(&sum, sum_of_exp, sizeof(float), hipMemcpyDeviceToHost);
    int eee;
    hipMemcpy(&eee, dev_elems, sizeof(int), hipMemcpyDeviceToHost);
    output_type z;
    if (data.activation_function == SOFTMAX) hipMemcpy(z, &dev_z[eee], 10*sizeof(float), hipMemcpyDeviceToHost);*/
    calc_a_and_dz<<<data.n_out.x, 1>>>(dev_z, dev_a, dev_dz, dev_elems, &dev_data->activation_function, sum_of_exp);
    /*output_type aa;
    if (data.activation_function == SOFTMAX) hipMemcpy(aa, &dev_a[eee], 10*sizeof(float), hipMemcpyDeviceToHost);*/
    hipDeviceSynchronize();
}

void fully_connected_layer::backprop(float * &delta, float* &activations, float* &derivative_z, int* elems) {
    float* dev_new_delta;
    hipMalloc((void**) &dev_new_delta, data.n_in.x*sizeof(float));
    set_to<<<data.n_in.x,1>>>(dev_new_delta, 0);

    if (!data.last_layer) {
        mult<<<data.n_out.x,1>>>(delta, derivative_z, elems);
    }

    hipDeviceSynchronize();

    add<<<data.n_out.x,1>>>(dev_biases_updt, delta);
    hipDeviceSynchronize();

    backprop_logic<<<data.n_out.x,data.n_in.x>>>(dev_weights_updt, delta, activations, dev_new_delta, dev_weights, &dev_data->n_in.x, elems);

    hipDeviceSynchronize();

    hipFree(delta);
    hipMalloc((void**) &delta, data.n_in.x*sizeof(float));
    hipMemcpy(dev_new_delta, delta, data.n_in.x*sizeof(float), hipMemcpyDeviceToDevice);
    hipFree(dev_new_delta);
}

void fully_connected_layer::update(hyperparams* dev_params) {
    // update velocities
    update_bias_vel<<<data.n_out.x,1>>>(dev_biases_vel, dev_biases_updt, dev_params);
    update_weights_vel<<<data.n_out.x*data.n_in.x,1>>>(dev_weights_vel, dev_weights_updt, dev_params);
    hipDeviceSynchronize();

    // update weights and biases
    add<<<data.n_out.x,1>>>(dev_biases, dev_biases_vel);
    update_weights<<<data.n_out.x*data.n_in.x,1>>>(dev_weights, dev_weights_vel, dev_params);
    hipDeviceSynchronize();

    set_to<<<data.n_out.x,1>>>(dev_biases_updt, 0);
    set_to<<<data.n_out.x*data.n_in.x,1>>>(dev_weights_updt, 0);
    hipDeviceSynchronize();
}

void fully_connected_layer::save(string filename) {
    ofstream file(filename, std::ios_base::app);

    file << LAYER_NUM_FULLY_CONNECTED << "//";
    file << data.activation_function << "//";
    file << data.n_out.x << "//";

    float* biases = new float [data.n_out.x];
    hipMemcpy(biases, dev_biases, data.n_out.x*sizeof(float), hipMemcpyDeviceToHost);
    for (int bias = 0; bias < data.n_out.x; bias++) file << biases[bias] << " ";
    delete[] biases;
    file << "//";

    float* biases_vel = new float [data.n_out.x];
    hipMemcpy(biases_vel, dev_biases, data.n_out.x*sizeof(float), hipMemcpyDeviceToHost);
    for (int bias_vel = 0; bias_vel < data.n_out.x; bias_vel++) file << biases_vel[bias_vel] << " ";
    delete[] biases_vel;
    file << "//";

    float* weights = new float [data.n_out.x*data.n_in.x];
    hipMemcpy(weights, dev_weights, data.n_out.x*data.n_in.x*sizeof(float), hipMemcpyDeviceToHost);
    for (int weight = 0; weight < data.n_out.x*data.n_in.x; weight++) file << weights[weight] << " ";
    delete[] weights;
    file << "//";

    float* weights_vel = new float [data.n_out.x*data.n_in.x];
    hipMemcpy(weights_vel, dev_weights_vel, data.n_out.x*data.n_in.x*sizeof(float), hipMemcpyDeviceToHost);
    for (int weight = 0; weight < data.n_out.x*data.n_in.x; weight++) file << weights_vel[weight] << " ";
    delete[] weights_vel;
    file << "\n";

    file.close();
}

void fully_connected_layer::clear() {
    hipFree(dev_weights);
    hipFree(dev_weights_vel);
    hipFree(dev_weights_updt);
    hipFree(dev_biases);
    hipFree(dev_biases_vel);
    hipFree(dev_biases_updt);
    hipFree(dev_data_previous);
    hipFree(dev_data);
}

/*void convolutional_layer::init(layer_data data, layer_data data_previous) {

    data.n_in = data_previous.n_out;
    data.n_out.x = (data.n_in.x - data.receptive_field_length + 1) / data.stride_length;
    data.n_out.y = (data.n_in.y - data.receptive_field_length + 1) / data.stride_length;

    this->data = data;
    this->data_previous = data_previous;

    weights_size = data.n_in.feature_maps * data.n_out.feature_maps * data.receptive_field_length * data.receptive_field_length;

    normal_distribution<float> distribution(0.0, 1.0 / sqrt(data.receptive_field_length * data.receptive_field_length));

    biases = new float[data.n_out.feature_maps];
    biasesVelocity = new float[data.n_out.feature_maps];
    for (int map = 0; map < data.n_out.feature_maps; map++) {
        biases[map] = distribution(generator);
        biasesVelocity[map] = 0;
    }

    weights = new float[weights_size];
    weightsVelocity = new float[weights_size];
    for (int previous_map = 0; previous_map < data.n_in.feature_maps; previous_map++) {
        for (int map = 0; map < data.n_out.feature_maps; map++) {
            for (int kernel_y = 0; kernel_y < data.receptive_field_length; kernel_y++) {
                for (int kernel_x = 0; kernel_x < data.receptive_field_length; kernel_x++) {
                    weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] = distribution(generator);
                    weightsVelocity[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] = 0;
                }
            }
        }
    }

    updateB = new float[data.n_out.feature_maps];
    updateW = new float[weights_size];
    for (int bias = 0; bias < data.n_out.feature_maps; bias++) updateB[bias] = 0;
    for (int weight = 0; weight < weights_size; weight++) updateW[weight] = 0;
}

void convolutional_layer::feedforward(float* a, float* dz, float* &new_a, float* &new_dz) {
    (void) dz;

    float* z = new float [data.n_out.feature_maps * data.n_out.y * data.n_out.x];
    for (int i = 0; i < data.n_out.feature_maps * data.n_out.y * data.n_out.x; i++) z[i] = 0;

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) {
                for (int previous_map = 0; previous_map < data.n_in.feature_maps; previous_map++) {
                    for (int kernel_y = 0; kernel_y < data.receptive_field_length; kernel_y++) {
                        for (int kernel_x = 0; kernel_x < data.receptive_field_length; kernel_x++) {
                            z[get_data_index(map, y, x, data)] +=
                                    weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] *
                                    a[get_data_index(previous_map, y * data.stride_length + kernel_y, x * data.stride_length + kernel_x, data_previous)];
                        }
                    }
                }
                z[get_data_index(map, y, x, data)] += biases[map];
                new_a[get_data_index(map, y, x, data)] = activation_function(z[get_data_index(map, y, x, data)], data.activation_function);
                new_dz[get_data_index(map, y, x, data)] = activation_function_prime(z[get_data_index(map, y, x, data)], data.activation_function);
            }
        }
    }

    delete[] z;
}

void convolutional_layer::backprop(float * &delta,
                                   float* &activations,
                                   float* &derivative_z, float * &new_delta) {

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) delta[get_data_index(map, y, x, data)] *= derivative_z[get_data_index(map, y, x, data)];
        }
    }

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) {
                updateB[map] += delta[get_data_index(map, y, x, data)];
                for (int previous_map = 0; previous_map < data.n_in.feature_maps; previous_map++) {
                    for (int kernel_y = 0; kernel_y < data.receptive_field_length; kernel_y++) {
                        for (int kernel_x = 0; kernel_x < data.receptive_field_length; kernel_x++) {
                            new_delta[get_data_index(previous_map, y * data.stride_length + kernel_y, x * data.stride_length +
                                                                                      kernel_x, data_previous)] +=
                                    delta[get_data_index(map, y, x, data)] * weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)];
                            updateW[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] +=
                                    activations[get_data_index(previous_map, y * data.stride_length + kernel_y,
                                            x * data.stride_length + kernel_x, data)] * delta[get_data_index(map, y, x, data)];
                        }
                    }
                }
            }
        }
    }
}

void convolutional_layer::update(hyperparams params) {

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        biasesVelocity[map] = params.momentum_coefficient * biasesVelocity[map] -
                              (params.convolutional_biases_learning_rate / params.mini_batch_size) * updateB[map];
        biases[map] += biasesVelocity[map];
    }

    for (int previous_map = 0; previous_map < data.n_in.feature_maps; previous_map++) {
        for (int map = 0; map < data.n_out.feature_maps; map++) {
            for (int kernel_y = 0; kernel_y < data.receptive_field_length; kernel_y++) {
                for (int kernel_x = 0; kernel_x < data.receptive_field_length; kernel_x++) {
                    weightsVelocity[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] =
                            params.momentum_coefficient * weightsVelocity[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] -
                            (params.convolutional_weights_learning_rate / params.mini_batch_size /
                             (data.n_out.x * data.n_out.y) *
                             data.stride_length * data.stride_length) * updateW[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)];
                    weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] = (1 -
                                                                      params.convolutional_weights_learning_rate /
                                                                      (data.n_out.x * data.n_out.y) *
                                                                      data.stride_length * data.stride_length *
                                                                      params.L2_regularization_term /
                                                                      params.training_data_size) *
                                                                     weights[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)] +
                                                                     weightsVelocity[get_convolutional_weights_index(previous_map, map, kernel_y, kernel_x, data)];
                }
            }
        }
    }

    for (int i = 0; i < data.n_out.feature_maps; i++) updateB[i] = 0;
    for (int i = 0; i < data.n_in.feature_maps * data.n_out.feature_maps * data.receptive_field_length * data.receptive_field_length; i++) updateW[i] = 0;
}

void convolutional_layer::save(string filename) {
    ofstream file(filename, std::ios_base::app);

    file << LAYER_NUM_CONVOLUTIONAL << "//";
    file << data.activation_function << "//";
    file << data.stride_length << " " << data.receptive_field_length << " " << data.n_out.feature_maps << "//";

    for (int bias = 0; bias < data.n_out.feature_maps; bias++) file << biases[bias] << " ";
    file << "//";
    for (int biasVel = 0; biasVel < data.n_out.feature_maps; biasVel++) file << biasesVelocity[biasVel] << " ";
    file << "//";
    for (int weight = 0; weight < weights_size; weight++) file << weights[weight] << " ";
    file << "//";
    for (int weightVel = 0; weightVel < weights_size; weightVel++) file << weightsVelocity[weightVel] << " ";
    file << "\n";

    file.close();
}

void convolutional_layer::clear() {
    delete[] weights;
    delete[] weightsVelocity;
    delete[] biases;
    delete[] biasesVelocity;
    delete[] updateW;
    delete[] updateB;
}

void max_pooling_layer::init(layer_data data, layer_data data_previous) {
    data.n_in = data_previous.n_out;
    this->data = data;
    this->data_previous = data_previous;
    this->data.n_out.x = data.n_in.x / data.summarized_region_length;
    this->data.n_out.y = data.n_in.y / data.summarized_region_length;
    this->data.n_out.feature_maps = data_previous.n_out.feature_maps;
}

void max_pooling_layer::feedforward(float* a, float* dz, float* &new_a, float* &new_dz) {
    (void) dz;

    for (int i = 0; i < data.n_out.feature_maps * data.n_out.y * data.n_out.x; i++) new_a[i] = numeric_limits<float>::lowest();

    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) {
                for (int kernel_y = 0; kernel_y < data.summarized_region_length; kernel_y++) {
                    for (int kernel_x = 0; kernel_x < data.summarized_region_length; kernel_x++) {
                        new_a[get_data_index(map, y, x, data)] = max(new_a[get_data_index(map, y, x, data)], a[get_data_index(map, y * data.summarized_region_length + kernel_y, x * data.summarized_region_length + kernel_x, data_previous)]);
                    }
                }
                new_dz[get_data_index(map, y, x, data)] = new_a[get_data_index(map, y, x, data)];
            }
        }
    }
}

void max_pooling_layer::backprop(float * &delta,
                                 float* &activations, float* &derivative_z, float * &new_delta) {
    const float epsilon = 1e-8;

    //cout << activations[get_data_index(data.n_out.feature_maps-1, (data.n_out.y-1)*data.summarized_region_length+data.summarized_region_length-1, (data.n_out.x-1)*data.summarized_region_length+data.summarized_region_length-1, data_previous)] << "sdfkjdslksfjlsf\n";
    for (int map = 0; map < data.n_out.feature_maps; map++) {
        for (int y = 0; y < data.n_out.y; y++) {
            for (int x = 0; x < data.n_out.x; x++) {
                for (int kernel_y = 0; kernel_y < data.summarized_region_length; kernel_y++) {
                    for (int kernel_x = 0; kernel_x < data.summarized_region_length; kernel_x++) {
                        int act = activations[get_data_index(map, y * data.summarized_region_length + kernel_y, x * data.summarized_region_length + kernel_x, data_previous)];
                        int dev = derivative_z[get_data_index(map, y, x, data)];
                        if (act < dev) swap(act, dev);
                        if (act - dev < epsilon) {
                            new_delta[get_data_index(map, y * data.summarized_region_length + kernel_y,
                                    x * data.summarized_region_length + kernel_x, data_previous)] = delta[get_data_index(map, y, x, data)];
                        }
                    }
                }
            }
        }
    }
}

void max_pooling_layer::update(hyperparams params) {
    (void) params;
}

void max_pooling_layer::save(string filename) {
    ofstream file(filename, std::ios_base::app);

    file << LAYER_NUM_MAX_POOLING << "//";
    file << data.summarized_region_length << "\n";

    file.close();
}

void max_pooling_layer::clear() {}
*/
void input_layer::init(layer_data data, layer_data data_previous) {
    this->data = data;
    (void) data_previous;
}

void input_layer::feedforward(float* &a, float* &dz, float* &dev_z, int* elems) {}

void input_layer::backprop(float * &delta,
                           float* &activations, float* &derivative_z, int* elems) {}

void input_layer::update(hyperparams* params) {}

void input_layer::save(string filename) {
    ofstream file(filename, std::ios_base::app);

    file << LAYER_NUM_INPUT << "//";
    file << data.n_out.x << " " << data.n_out.y << "\n";

    file.close();
}

void input_layer::clear() {}