#include "hip/hip_runtime.h"
#include "includes.h"

// sigmoid function and its derivative
float sigmoid(float x) {
    return 1.0 / (1.0 + exp(-x));
}
float sigmoidPrime(float x) {
    return (sigmoid(x)*(1-sigmoid(x)));
}

float relu(float x){
    return max(x, 0.0f);
}

float reluPrime(float x){
    if (x > 0) return 1;
    return (float)0;
}

// cross entropy cost function
float crossEntropyPrime(float output_activation, float y) {
    return (output_activation-y);
}

int get_convolutional_weights_index(int previous_map, int map, int y, int x, layer_data &data) {
    return
            previous_map * (data.n_out.feature_maps * data.receptive_field_length * data.receptive_field_length)
            + map * (data.receptive_field_length * data.receptive_field_length)
            + y * (data.receptive_field_length)
            + x;
}

int get_data_index(int map, int y, int x, layer_data &data) {
    return
            map * (data.n_out.x * data.n_out.y)
            + y * (data.n_out.x)
            + x;
}

int get_fully_connected_weight_index(int neuron, int previous_neuron, int data_n_in) {
    return neuron*data_n_in+previous_neuron;
}

__device__ int get_fully_connected_weight_index_dev (int neuron, int previous_neuron, int data_n_in) {
    return neuron*data_n_in+previous_neuron;
}

// load data
pair<data_point*, int> load_data(string filename) {
    // loads data from csv file of form label, pixel1, pixel2, pixel3, ..., pixel784
    ifstream file;
    string line;

    file.open(filename);

    // how many lines there are in the file
    int dataPoints = 0;
    while (getline(file, line)) {
        dataPoints++;
    }
    file.close();

    file.open(filename);

    data_point *data = new data_point[dataPoints];
    int lineIndex = 0;

    while (getline(file, line)) {
        stringstream ss(line);

        for (int i = 0; i < 10; i++) data[lineIndex].second[i] = 0;
        for (int i = 0; i < 28 * 28; i++) data[lineIndex].first[i] = 0;

        int label = -1;
        int i = 0;
        while (ss.good()) {
            string substr;
            getline(ss, substr, ' ');
            if (label == -1) {
                label = stoi(substr);
            } else {
                if (i == 28 * 28) break;
                data[lineIndex].first[i] = atof(substr.c_str());
                i++;
            }
        }
        data[lineIndex].second[label] = 1;
        lineIndex++;
    }
    cerr << dataPoints << " data loaded from " + filename + "\n";
    file.close();
    return {data, dataPoints};
}

hyperparams get_params() {
    hyperparams params;

    params.mini_batch_size = 16;
    params.epochs = 5;

    params.fully_connected_weights_learning_rate = 1.2*0.017599067515299563;
    params.fully_connected_biases_learning_rate = 1.2*0.041000786959874205;
    params.convolutional_weights_learning_rate = 1.2*1.0075;
    params.convolutional_biases_learning_rate = 1.2*0.011;

    params.L2_regularization_term = 0;
    params.momentum_coefficient = 0;

    return params;
}

void clear_data(data_point *data) {
    delete[] data;
}

__global__ void addWeights (float* a, float* weights, float* z, int* data_n_in, int* offset) {
    int neuron = blockIdx.x;
    int previous_neuron = threadIdx.x;
    atomicAdd(&z[(*offset)+neuron], weights[get_fully_connected_weight_index_dev(neuron, previous_neuron, *data_n_in)] * a[(*offset)-(*data_n_in)+previous_neuron]);
}

__global__ void getNewA (float* z, float* biases, float* new_a, float* new_dz, int* offset) {
    int neuron = blockIdx.x;
    z[(*offset)+neuron] += biases[neuron];
    // TODO : actually use the activation function
    if (z[(*offset)+neuron] >= 0) {
        new_a[(*offset)+neuron] = z[(*offset)+neuron];
        new_dz[(*offset)+neuron] = 1;
    } else {
        new_a[(*offset)+neuron] = 0;
        new_dz[(*offset)+neuron] = 0;
    }
}

__global__ void backprop_logic (float* dev_weights_upt, float* dev_delta, float* dev_activations, float* dev_new_delta, float* dev_weights, int* data_n_in_x, int *offset) {
    int neuron = blockIdx.x;
    int previous_neuron = threadIdx.x;
    atomicAdd(&dev_weights_upt[get_fully_connected_weight_index_dev(neuron, previous_neuron, *data_n_in_x)], dev_delta[neuron] * dev_activations[(*offset)-(*data_n_in_x)+previous_neuron]);
    atomicAdd(&dev_new_delta[previous_neuron], dev_delta[neuron] * dev_weights[get_fully_connected_weight_index_dev(neuron, previous_neuron, *data_n_in_x)]);
}

__global__ void update_bias_vel (float* biases_vel, float* biases_updt, hyperparams* params) {
    int neuron = blockIdx.x;
    biases_vel[neuron] = params->momentum_coefficient * biases_vel[neuron] -
                             (params->fully_connected_biases_learning_rate / params->mini_batch_size) *
                             biases_updt[neuron];
}

__global__ void update_weights_vel (float* weights_vel, float* weights_updt, hyperparams* params) {
    int weight = blockIdx.x;
    weights_vel[weight] =
            params->momentum_coefficient * weights_updt[weight] -
            (params->fully_connected_weights_learning_rate / params->mini_batch_size) *
            weights_updt[weight];
}

__global__ void update_weights (float* weights, float* weights_vel, hyperparams* params) {
    int weight = blockIdx.x;
    weights[weight] = (1 - params->fully_connected_weights_learning_rate * params->L2_regularization_term
                        / params->training_data_size) * weights[weight] + weights_vel[weight];
}

__global__ void set_to (float *vec, float value) {
    int index = blockIdx.x;
    vec[index] = value;
}

__device__ int sqrt(int num) {
    int l = 0;
    int r = num;
    while (l+1 != r) {
        int m = (l+r)/2;
        if (m*m > num) r = m;
        else l = m;
    }
    return l;
}

__global__ void set_to_random (float *vec, int *data_n_in_x) {
    int index = blockIdx.x;

    hiprandState state;
    hiprand_init(clock64(), index, 0, &state);
    vec[index] = hiprand_normal(&state)/sqrt(*data_n_in_x);
    //vec[index] = 0;
}

__global__ void add (float *vec_a, float *vec_b) {
    int index = blockIdx.x;
    vec_a[index] += vec_b[index];
}

__global__ void mult (float *vec_a, float *vec_b, int *offset_b) {
    int index = blockIdx.x;
    vec_a[index] *= vec_b[index+(*offset_b)];
}