#include "hip/hip_runtime.h"
#include "includes.h"

// sigmoid function and its derivative
inline __device__ float sigmoid(float x) {
    return 1.0 / (1.0 + expf(-x));
}

inline __device__ float sigmoid_prime(float x) {
    return (sigmoid(x)*(1-sigmoid(x)));
}

inline __device__ float relu(float x){
    return max(x, 0.0f);
}

inline __device__ float relu_prime(float x){
    if (x > 0) return 1.0f;
    return 0.0f;
}

inline __device__ float softmax(float x, float sum_of_exp) {
    return expf(x)/sum_of_exp;
}

inline __device__ float softmax_prime(float x, float sum_of_exp) {
    return (softmax(x, sum_of_exp)*(1-softmax(x, sum_of_exp)));
}

inline __device__ float cross_entropy_prime(float out_net, float out_cor) {
    return (out_net-out_cor);
}

inline __device__ float activation_function(float x, int activation_func, float sum_of_exp) {
    if (activation_func == SIGMOID) return sigmoid(x);
    else if (activation_func == RELU) return relu(x);
    else if (activation_func == SOFTMAX) return softmax(x, sum_of_exp);
    else return 0;
}

inline __device__ float activation_function_prime(float x, int activation_func, float sum_of_exp) {
    if (activation_func == SIGMOID) return sigmoid_prime(x);
    else if (activation_func == RELU) return relu_prime(x);
    else if (activation_func == SOFTMAX) return softmax_prime(x, sum_of_exp);
    else return 0;
}

inline __device__ float cost_function_prime(float out_net, float out_cor, int cost_function) {
    if (cost_function == CROSSENTROPY) return cross_entropy_prime(out_net, out_cor);
    else return 0;
}

int get_convolutional_weights_index(int previous_map, int map, int y, int x, layer_data &data) {
    return
            previous_map * (data.n_out.feature_maps * data.receptive_field_length * data.receptive_field_length)
            + map * (data.receptive_field_length * data.receptive_field_length)
            + y * (data.receptive_field_length)
            + x;
}

int get_data_index(int map, int y, int x, layer_data &data) {
    return
            map * (data.n_out.x * data.n_out.y)
            + y * (data.n_out.x)
            + x;
}

inline __device__ int get_fully_connected_weight_index_dev (int neuron, int previous_neuron, int data_n_in) {
    return neuron*data_n_in+previous_neuron;
}

// load data
pair<vector<pair<float*,float*>>, int> load_data(string filename) {
    // loads data from csv file of form label, pixel1, pixel2, pixel3, ..., pixel784
    ifstream file;
    string line;

    file.open(filename);

    // how many lines there are in the file
    int dataPoints = 0;
    while (getline(file, line)) {
        dataPoints++;
    }

    file.clear(); // Reset stream state
    file.seekg(0); // Move cursor back to beginning

    int lineIndex = 0;
    vector<pair<float*,float*>> data (dataPoints, {nullptr, nullptr});

    while (getline(file, line)) {
        stringstream ss(line);
        float* data_in = new float [INPUT_NEURONS];
        float* data_out = new float [OUTPUT_NEURONS];

        for (int i = 0; i < INPUT_NEURONS; i++) data_in[i] = 0;
        for (int i = 0; i < OUTPUT_NEURONS; i++) data_out[i] = 0;

        int label = -1;
        int i = 0;
        while (ss.good()) {
            string substr;
            getline(ss, substr, ' ');
            if (label == -1) {
                label = stoi(substr);
            } else {
                if (i == INPUT_NEURONS) break;
                data_in[i] = atof(substr.c_str());
                i++;
            }
        }
        data_out[label] = 1;

        float* dev_data_in;
        float* dev_data_out;
        hipMalloc((void**) &dev_data_in, INPUT_NEURONS*sizeof(float));
        hipMalloc((void**) &dev_data_out, OUTPUT_NEURONS*sizeof(float));
        hipMemcpy(dev_data_in, data_in, INPUT_NEURONS*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_data_out, data_out, OUTPUT_NEURONS*sizeof(float), hipMemcpyHostToDevice);
        data[lineIndex] = {dev_data_in, dev_data_out};

        lineIndex++;

        delete [] data_in;
        delete [] data_out;
    }

    cerr << dataPoints << " data loaded from " + filename + "\n";
    file.close();
    return {data, dataPoints};
}

hyperparams get_params() {
    hyperparams params;

    params.mini_batch_size = 16;
    params.epochs = 5;

    params.fully_connected_weights_learning_rate = 1.2*0.017599067515299563;
    params.fully_connected_biases_learning_rate = 1.2*0.041000786959874205;
    params.convolutional_weights_learning_rate = 1.2*1.0075;
    params.convolutional_biases_learning_rate = 1.2*0.011;

    params.L2_regularization_term = 0;
    params.momentum_coefficient = 0;

    params.cost = CROSSENTROPY;

    return params;
}

void clear_data(vector<pair<float*,float*>> & data) {
    for (int data_point = 0; data_point < (int)data.size(); data_point++) {
        hipFree(data[data_point].first);
        hipFree(data[data_point].second);
    }
}

__global__ void calc_a_and_dz (float* z, float* new_a, float* new_dz, int* activation_func, float* sum_of_exp) {
    int neuron = blockIdx.x;

    new_a[neuron] = activation_function(z[ + neuron], *activation_func, *sum_of_exp);
    new_dz[neuron] = activation_function_prime(z[neuron], *activation_func, *sum_of_exp);
}

__global__ void set_delta (float* delta, float* activations, float* out, int* cost_func) {
    int neuron = blockIdx.x;
    delta[neuron] = cost_function_prime(activations[neuron], out[neuron], *cost_func);
}

__global__ void backprop_logic (float* dev_weights_upt, float* dev_delta, float* dev_activations, float* dev_new_delta, float* dev_weights, int* data_n_in_x) {
    int neuron = blockIdx.x;
    int previous_neuron = threadIdx.x;
    atomicAdd(&dev_weights_upt[get_fully_connected_weight_index_dev(neuron, previous_neuron, *data_n_in_x)], dev_delta[neuron] * dev_activations[previous_neuron]);
    atomicAdd(&dev_new_delta[previous_neuron], dev_delta[neuron] * dev_weights[get_fully_connected_weight_index_dev(neuron, previous_neuron, *data_n_in_x)]);
}

__global__ void update_bias_vel (float* biases_vel, float* biases_updt, hyperparams* params) {
    int neuron = blockIdx.x;
    biases_vel[neuron] = params->momentum_coefficient * biases_vel[neuron] -
                             (params->fully_connected_biases_learning_rate / params->mini_batch_size) *
                             biases_updt[neuron];
}

__global__ void update_weights_vel (float* weights_vel, float* weights_updt, hyperparams* params) {
    int weight = blockIdx.x;
    weights_vel[weight] =
            params->momentum_coefficient * weights_updt[weight] -
            (params->fully_connected_weights_learning_rate / params->mini_batch_size) *
            weights_updt[weight];
}

__global__ void update_weights (float* weights, float* weights_vel, hyperparams* params) {
    int weight = blockIdx.x;
    weights[weight] = (1 - params->fully_connected_weights_learning_rate * params->L2_regularization_term
                        / params->training_data_size) * weights[weight] + weights_vel[weight];
}

__global__ void set_to (float *vec, float value) {
    int index = blockIdx.x;
    vec[index] = value;
}

__global__ void set_to_random (float *vec, float *stddev) {
    int index = blockIdx.x;

    hiprandState state;
    hiprand_init(clock64(), index, 0, &state);
    vec[index] = hiprand_normal(&state)*(*stddev);
}

__global__ void add (float *vec_a, float *vec_b) {
    int index = blockIdx.x;
    vec_a[index] += vec_b[index];
}

__global__ void mult (float *vec_a, float *vec_b) {
    // TODO: here
    int bid = blockIdx.x; // previous neuron
    int tid = threadIdx.x;
    int index = tid*gridDim.x+bid;
    // neuron*data_n_in + previous_neuron
    vec_a[index] *= vec_b[bid];
}
// weights[get_fully_connected_weight_index_dev(neuron, previous_neuron, *data_n_in)] * a[(*offset)-(*data_n_in)+previous_neuron]

__global__ void calc_exp (float* res, float* vec, int* max_id) {
    int index = blockIdx.x;
    res[index] = expf(vec[index]-vec[*max_id]);
}

__global__ void find_max (float* vec, int* id, int* size) {
    int index = blockIdx.x;
    (*id) = 0;
    for (int i = 0; i < (*size); i++) {
        if (vec[index+i] > vec[index+(*id)]) (*id) = i;
    }
}

inline __device__ void reduce_last_warp(volatile float* sum, int ind, int block_size) {
    if (block_size > 32) {
        if (ind < block_size - 32 && ind < 32) sum[ind] += sum[ind + 32];
    }
    if (block_size > 16) {
        if (ind < block_size - 16 && ind < 16) sum[ind] += sum[ind + 16];
    }
    if (block_size > 8) {
        if (ind < block_size - 8 && ind < 8) sum[ind] += sum[ind + 8];
    }
    if (block_size > 4) {
        if (ind < block_size - 4 && ind < 4) sum[ind] += sum[ind + 4];
    }
    if (block_size > 2) {
        if (ind < block_size - 2 && ind < 2) sum[ind] += sum[ind + 2];
    }
    if (block_size > 1) {
        if (ind < block_size - 1 && ind < 1) sum[ind] += sum[ind + 1];
    }
}

__global__ void reduce(float* input, float* res, int* size, int* block_size_ptr) {
    const int block_size = *block_size_ptr;
    extern __shared__ float sum[];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int add = block_size;
    sum[tid] = input[bid * block_size + tid]; // TODO: this can use like a lambda or whatever; this is the only thing that actually needs to be changed
    while (tid + add < *size) {
        sum[tid] += input[bid * block_size + tid + add];
        add += block_size;
    }
    __syncthreads();

    if (block_size > 512) {
        if (tid < block_size - 512) sum[tid] += sum[tid + 512];
        __syncthreads();
    }
    if (block_size > 256) {
        if (tid < block_size - 256 && tid < 256) sum[tid] += sum[tid + 256];
        __syncthreads();
    }
    if (block_size > 128) {
        if (tid < block_size - 128 && tid < 128) sum[tid] += sum[tid + 128];
        __syncthreads();
    }
    if (block_size > 64) {
        if (tid < block_size - 64 && tid < 64) sum[tid] += sum[tid + 64];
        __syncthreads();
    }

    if (tid < 32) reduce_last_warp(sum, tid, block_size);
    if (tid == 0) res[bid] += sum[tid];
}