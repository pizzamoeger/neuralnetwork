#include "includes.h"

void Network::init(layer_data* layers, int L, hyperparams params) {

    this->L = L;
    this->params = params;
    hipMalloc((void**) &dev_params, sizeof(hyperparams));
    hipMemcpy(dev_params, &params, sizeof(hyperparams), hipMemcpyHostToDevice);
    this->layers = new unique_ptr<layer>[L];

    // initialize layers
    for (int l = 0; l < L; l++) {
        unique_ptr<layer> new_layer = nullptr;
        switch (layers[l].type) {
            case LAYER_NUM_INPUT:
                new_layer = make_unique<input_layer>();
                break;
            case LAYER_NUM_CONVOLUTIONAL:
                //new_layer = make_unique<convolutional_layer>();
                break;
            case LAYER_NUM_MAX_POOLING:
                //new_layer = make_unique<max_pooling_layer>();
                break;
            case LAYER_NUM_FULLY_CONNECTED:
                new_layer = make_unique<fully_connected_layer>();
                break;
        }
        layer_data previous_data;
        if (l > 0) previous_data = this->layers[l - 1]->data;
        new_layer->init(layers[l], previous_data);
        this->layers[l] = move(new_layer);
    }
}

void Network::feedforward(float* a, float* dev_activations, float* dev_derivatives_z) {

    hipMemcpy(dev_activations, a, INPUT_NEURONS*sizeof(float), hipMemcpyDeviceToDevice);

    for (int l = 1; l < L; l++) {
        layers[l]->feedforward(dev_activations, dev_derivatives_z);
    }

}

pair<int,int> Network::evaluate(vector<pair<float*,float*>> test_data, int test_data_size) {
    auto start = chrono::high_resolution_clock::now();

    int* dev_correct;
    hipMalloc((void**) &dev_correct, sizeof(int));
    hipMemcpy(dev_correct, zero_pointer, sizeof(int), hipMemcpyDeviceToDevice);

    int elems = layers[L-1]->data.elems+OUTPUT_NEURONS;
    float* activations;
    float* derivatives_z;

    hipMalloc((void**) &activations, elems*sizeof(float));
    hipMalloc((void**) &derivatives_z, elems*sizeof(float));

    for (int k = 0; k < (int) test_data_size; k++) {
        feedforward(test_data[k].first, activations, derivatives_z);
        hipDeviceSynchronize();
        eval<<<1,1>>>(test_data[k].second, &activations[layers[L-1]->data.elems], dev_correct, &layers[L-1]->dev_data->n_out.x);
    }
    hipDeviceSynchronize();

    int correct;
    hipMemcpy(&correct, dev_correct, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(derivatives_z);
    hipFree(activations);
    hipFree(dev_correct);
    auto end = chrono::high_resolution_clock::now();
    return {correct, chrono::duration_cast<chrono::milliseconds>(end - start).count()};
}

void Network::SGD(vector<pair<float*,float*>> training_data, vector<pair<float*,float*>> test_data) {

    auto ev = evaluate(test_data, params.test_data_size);
    auto correct = ev.first;
    auto durationEvaluate = ev.second;
    cerr << "0 Accuracy: " << (float) correct / params.test_data_size << " evaluated in " << durationEvaluate << "ms\n";

    for (int i = 0; i < params.epochs; i++) {
        // time the epoch
        auto start = chrono::high_resolution_clock::now();

        cerr << i+1 << " ";

        // obtain a time-based seed
        unsigned seed = chrono::system_clock::now().time_since_epoch().count();
        shuffle(training_data.begin(), training_data.end(), default_random_engine(seed));

        // create mini batches and update them
        vector<pair<float*,float*>> mini_batch (params.mini_batch_size, {nullptr, nullptr});
        for (int j = 0; j < params.training_data_size / params.mini_batch_size; j++) {
            for (int k = 0; k < params.mini_batch_size; k++) {
                mini_batch[k].first = training_data[j * params.mini_batch_size + k].first;
                mini_batch[k].second = training_data[j * params.mini_batch_size + k].second;
            }
            update_mini_batch(mini_batch);
        }

        // end the timer
        auto end = chrono::high_resolution_clock::now();
        auto durationTrain = chrono::duration_cast<chrono::milliseconds>(end - start).count();

        // evaluate the network
        ev = evaluate(test_data, params.test_data_size);
        correct = ev.first;
        durationEvaluate = ev.second;

        cerr << "Accuracy: " << (float) correct / params.test_data_size << ", trained in " << durationTrain << "ms, evaluated in " << durationEvaluate << "ms\n";

        // reduce learning rate
	    if (i < 100) {
            params.fully_connected_biases_learning_rate -= params.fcBRed;
            params.fully_connected_weights_learning_rate -= params.fcWRed;
            params.convolutional_biases_learning_rate -= params.convBRed;
            params.convolutional_weights_learning_rate -= params.convWRed;
            //hipFree(dev_params);
            //hipMalloc((void**) &dev_params, sizeof(hyperparams));
            hipMemcpy(dev_params, &params, sizeof(hyperparams), hipMemcpyHostToDevice);
        }
    }
}

void Network::update_mini_batch(vector<pair<float*,float*>> mini_batch) {

    for (int num = 0; num < params.mini_batch_size; num++) {
        backprop(mini_batch[num].first, mini_batch[num].second);
    }

    // update velocities
    for (int i = 1; i < L; i++) layers[i]->update(dev_params);
}

void Network::backprop(float* in, float* out) {
    // feedfoward

    int elems = layers[L-1]->data.elems+OUTPUT_NEURONS;

    float* activations;
    float* derivatives_z;

    hipMalloc((void**) &activations, elems*sizeof(float));
    hipMalloc((void**) &derivatives_z, elems*sizeof(float));
    feedforward(in, activations, derivatives_z);

    elems = 0;
    for (int l = 0; l < L-1; l++) elems += layers[l]->data.n_out.x*layers[l]->data.n_out.y*layers[l]->data.n_out.feature_maps;
    int* dev_elems;
    hipMalloc((void**) &dev_elems, sizeof(int));
    hipMemcpy(dev_elems, &elems, sizeof(int), hipMemcpyHostToDevice);

    // backpropagate
    float* delta;
    hipMalloc((void**) &delta, OUTPUT_NEURONS*sizeof(float));
    set_to<<<OUTPUT_NEURONS, 1>>> (delta, 0);
    hipDeviceSynchronize();

    set_delta<<<OUTPUT_NEURONS,1>>> (delta, &activations[elems], out, &dev_params->cost);

    for (int l = L - 1; l >= 1; l--) {
        layers[l]->backprop(delta, activations, derivatives_z, dev_elems);
        elems -= layers[l-1]->data.n_out.x*layers[l-1]->data.n_out.y*layers[l-1]->data.n_out.feature_maps;
        hipMemcpy(dev_elems, &elems, sizeof(int), hipMemcpyHostToDevice);
    }

    // clean
    hipFree(dev_elems);
    hipFree(delta);
    hipFree(activations);
    hipFree(derivatives_z);
}

void Network::save(string filename) {
    ofstream file(filename);
    file << L << "\n";
    file.close();

    for (int l = 0; l < L; l++) layers[l]->save(filename);
}

void Network::clear() {
    for (int l = 0; l < L; l++) layers[l]->clear();

    hipFree(dev_params);
    delete[] layers;
}